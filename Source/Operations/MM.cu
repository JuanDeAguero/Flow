#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArray.h"

#define TILE_SIZE 32

using namespace std;

__global__
void MM_Kernel( float* arr1, float* arr2, float* result, int arr1Rows, int arr1Cols, int arr2Cols )
{
    __shared__ float arr1_s[TILE_SIZE][TILE_SIZE];
    __shared__ float arr2_s[TILE_SIZE][TILE_SIZE];
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + threadY;
    int col = blockIdx.x * TILE_SIZE + threadX;
    float sum = 0.0f;
    for ( int i = 0; i < ceil( arr1Cols / float(TILE_SIZE) ); i++ )
    {
        if ( row < arr1Rows && ( i * TILE_SIZE + threadX ) < arr1Cols )
            arr1_s[threadY][threadX] = arr1[ row * arr1Cols + i * TILE_SIZE + threadX ];
        else arr1_s[threadY][threadX] = 0.0f;
        if ( ( i * TILE_SIZE + threadY ) < arr1Cols && col < arr2Cols )
            arr2_s[threadY][threadX] = arr2[ ( i * TILE_SIZE + threadY ) * arr2Cols + col ];
        else arr2_s[threadY][threadX] = 0.0f;
        __syncthreads();
        for ( int k = 0; k < TILE_SIZE; k++ )
            sum += arr1_s[threadY][k] * arr2_s[k][threadX];
        __syncthreads();
    }
    if ( row < arr1Rows && col < arr2Cols )
        result[ row * arr2Cols + col ] = sum;
}

pair< vector<int>, float* > Flow::MMRaw( pair< vector<int>, float* > arr1,
    pair< vector<int>, float* > arr2 )
{
    int arr1Rows = arr1.first[0];
    int arr1Cols = arr1.first[1];
    int arr2Cols = arr2.first[1];
    float* result_d;
    hipMalloc( (void**)&result_d, arr1Rows * arr2Cols * sizeof(float) );
    dim3 dimGrid( ceil( arr2Cols / float(TILE_SIZE) ), ceil( arr1Rows / float(TILE_SIZE) ), 1 );
    dim3 dimBlock( TILE_SIZE, TILE_SIZE, 1 );
    MM_Kernel<<< dimGrid, dimBlock >>>( arr1.second, arr2.second, result_d, arr1Rows, arr1Cols,
        arr2Cols );
    hipDeviceSynchronize();
    return { { arr1Rows, arr2Cols }, result_d };
}

pair< vector<int>, float* > Flow::MMRaw( NARRAY arr1, NARRAY arr2 )
{
    return MMRaw( { arr1->GetShape(), arr1->GetData() }, { arr2->GetShape(), arr2->GetData() } );
}

pair< vector<int>, float* > Flow::MMRaw( pair< vector<int>, float* > arr1, NARRAY arr2 )
{
    return MMRaw( arr1, { arr2->GetShape(), arr2->GetData() } );
}

pair< vector<int>, float* > Flow::MMRaw( NARRAY arr1, pair< vector<int>, float* > arr2 )
{
    return MMRaw( { arr1->GetShape(), arr1->GetData() }, arr2 );
}

NARRAY Flow::MM( NARRAY arr1, NARRAY arr2 )
{
    auto mm = MMRaw( arr1, arr2 );
    return Create( mm.first, mm.second, { arr1, arr2 }, NArray::Operation::MM );
}

void Flow::NArray::BackwardMM()
{
    auto transpose1 = TransposeRaw( Operands[0], 0, 1 );
    auto transpose2 = TransposeRaw( Operands[1], 0, 1 );
    auto grad1 = MMRaw( Gradient, transpose2 );
    auto grad2 = MMRaw( transpose1, Gradient );
    hipMemcpy( Operands[0]->Gradient->Data, grad1.second,
        SizeFromShape(grad1.first) * sizeof(float), hipMemcpyDeviceToDevice );
    hipMemcpy( Operands[1]->Gradient->Data, grad2.second,
        SizeFromShape(grad2.first) * sizeof(float), hipMemcpyDeviceToDevice );
    hipFree(grad1.second);
    hipFree(grad2.second);
    hipFree(transpose1.second);
    hipFree(transpose2.second);
}