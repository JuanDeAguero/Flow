#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArray.h"
    
NARRAY Flow::Unsqueeze( NARRAY arr, int dim )
{
    vector<int> resultShape = arr->GetShape();
    resultShape.insert( resultShape.begin() + dim, 1 );
    float* result_d;
    hipMalloc( (void**)&result_d, SizeFromShape(arr->GetShape()) * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), SizeFromShape(arr->GetShape()) * sizeof(float),
        hipMemcpyDeviceToDevice );
    return Create( resultShape, result_d, { arr }, NArray::Operation::UNSQUEEZE );
}

__global__
void BackwardUnsqueeze_Kernel( float* gradient, float* operandGradient )
{
    int i = blockIdx.x;
    atomicAdd( &operandGradient[i], gradient[i] );
}

void Flow::NArray::BackwardUnsqueeze()
{
    int n = SizeFromShape(Shape);
    BackwardUnsqueeze_Kernel<<< n, 1 >>>( Gradient->GetData(),
        Operands[0]->GetGradient()->GetData() );
    hipDeviceSynchronize();
}