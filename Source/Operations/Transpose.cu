#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

namespace Flow
{
    __global__
    void Transpose_Kernel( float* arr, int* arrShape, int arrShapeSize, int firstDim, int secondDim, float* result, int* resultShape, int resultShapeSize )
    {
        int i = blockIdx.x;
        int multiIndex[10];
        FlatToMultiIndex_Device( i, arrShape, arrShapeSize, multiIndex );
        int temp = multiIndex[firstDim];
        multiIndex[firstDim] = multiIndex[secondDim];
        multiIndex[secondDim] = temp;
        int flatIndex = MultiToFlatIndex_Device( multiIndex, resultShape, resultShapeSize );
        result[flatIndex] = arr[i];
    }

    __host__
    NArrayCore* Transpose( NArrayCore* arr, int firstDim, int secondDim )
    {
        return nullptr;

        /*auto start = chrono::high_resolution_clock::now();

        vector<int> resultShape = arr->GetShape();
        int temp = resultShape[firstDim];
        resultShape[firstDim] = resultShape[secondDim];
        resultShape[secondDim] = temp;
        //vector<float> resultData( SizeFromShape(arr->GetShape()), 0 );
        int n = SizeFromShape(arr->GetShape());
        //float* arr_d = HostToDeviceArr(arr);
        int* arrshape_d = HostToDeviceVec<int>(arr->GetShape());

        float* result_d;
        hipMalloc( (void**)&result_d, SizeFromShape(arr->GetShape()) * sizeof(float) );

        int* resultshape_d = HostToDeviceVec<int>(resultShape);
        Transpose_Kernel<<< n, 1 >>>( arr->DeviceData, arrshape_d, arr->GetShape().size(), firstDim, secondDim, result_d, resultshape_d, resultShape.size() );
        //hipMemcpy( resultData.data(), result_d, resultData.size() * sizeof(float), hipMemcpyDeviceToHost );

        auto end = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::nanoseconds>( end - start );
        //Print( to_string(duration.count()) + " Transpose" );

        NArrayCore* result = new NArrayCore( resultShape, {}, { arr }, NArrayCore::Operation::TRANSPOSE );
        result->DeviceData = result_d;
        return result;*/
    }
}

void Flow::NArrayCore::BackwardTranspose()
{

}