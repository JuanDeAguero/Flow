#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Transpose_Kernel( float* arr, int* arrShape, int arrShapeSize, int firstDim, int secondDim, float* result, int* resultShape, int resultShapeSize )
{
    int i = blockIdx.x;
    int multiIndex[10];
    Flow::FlatToMultiIndex_Device( i, arrShape, arrShapeSize, multiIndex );
    int temp = multiIndex[firstDim];
    multiIndex[firstDim] = multiIndex[secondDim];
    multiIndex[secondDim] = temp;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, resultShape, resultShapeSize );
    result[flatIndex] = arr[i];
}

Flow::NArrayCore* Flow::Transpose( NArrayCore* arr, int firstDim, int secondDim )
{
    int n = SizeFromShape(arr->GetShape());
    vector<int> resultShape = arr->GetShape();
    int temp = resultShape[firstDim];
    resultShape[firstDim] = resultShape[secondDim];
    resultShape[secondDim] = temp;
    int* arrShape_d;
    float* result_d;
    int* resultShape_d;
    hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
    hipMalloc( (void**)&result_d, SizeFromShape(arr->GetShape()) * sizeof(float) );
    hipMalloc( (void**)&resultShape_d, resultShape.size() * sizeof(int) );
    hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( resultShape_d, resultShape.data(), resultShape.size() * sizeof(int), hipMemcpyHostToDevice );
    Transpose_Kernel<<< n, 1 >>>( arr->GetData(), arrShape_d, arr->GetShape().size(), firstDim, secondDim, result_d, resultShape_d, resultShape.size() );
    return new NArrayCore( resultShape, result_d, { arr }, NArrayCore::Operation::TRANSPOSE );
}

void Flow::NArrayCore::BackwardTranspose()
{

}