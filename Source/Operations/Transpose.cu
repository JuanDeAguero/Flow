#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <stdexcept>

#include "CUDA.cuh"
#include "Flow/NArray.h"

using namespace std;

__global__
void Transpose_Kernel( float* arr, int* arrShape, int arrShapeSize, int firstDim, int secondDim,
    float* result, int* resultShape, int resultShapeSize )
{
    int i = blockIdx.x;
    int multiIndex[MAX_DIMS];
    Flow::FlatToMultiIndex_Device( i, arrShape, arrShapeSize, multiIndex );
    int temp = multiIndex[firstDim];
    multiIndex[firstDim] = multiIndex[secondDim];
    multiIndex[secondDim] = temp;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, resultShape, resultShapeSize );
    result[flatIndex] = arr[i];
}

pair< vector<int>, float* > Flow::TransposeRaw( NARRAY arr, int firstDim, int secondDim )
{
    int n = SizeFromShape(arr->GetShape());
    vector<int> resultShape = arr->GetShape();
    int temp = resultShape[firstDim];
    resultShape[firstDim] = resultShape[secondDim];
    resultShape[secondDim] = temp;
    int* arrShape_d;
    float* result_d;
    int* resultShape_d;
    hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
    hipMalloc( (void**)&result_d, SizeFromShape(arr->GetShape()) * sizeof(float) );
    hipMalloc( (void**)&resultShape_d, resultShape.size() * sizeof(int) );
    hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( resultShape_d, resultShape.data(), resultShape.size() * sizeof(int),
        hipMemcpyHostToDevice );
    Transpose_Kernel<<< n, 1 >>>( arr->GetData(), arrShape_d, arr->GetShape().size(), firstDim,
        secondDim, result_d, resultShape_d, resultShape.size() );
    hipDeviceSynchronize();
    hipFree(arrShape_d);
    hipFree(resultShape_d);
    return { resultShape, result_d };
}

NARRAY Flow::Transpose( NARRAY arr, int firstDim, int secondDim )
{
    auto transpose = TransposeRaw( arr, firstDim, secondDim );
    return Create( transpose.first, transpose.second, { arr }, NArray::Operation::TRANSPOSE );
}

void Flow::NArray::BackwardTranspose()
{
    throw runtime_error("Backward transpose not implemented!");
}