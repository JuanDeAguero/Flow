#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void ElementWise_Kernel( float* arr1, float* arr2, float* result, int op )
{
    int i = blockIdx.x;
    switch (op)
    {
        case 1: result[i] = arr1[i] + arr2[i]; break;
        case 2: result[i] = arr1[i] * arr2[i]; break;
    }
}
    
namespace Flow
{
    __host__
    void ElementWise_CUDA( NArrayCore* arr1, NArrayCore* arr2, NArrayCore* result, NArrayCore::Operation op )
    {
        int n = arr1->Get().size();
        float* arr1_d;
        float* arr2_d;
        float* result_d;
        hipMalloc( (void**)&arr1_d, n * sizeof(float) );
        hipMalloc( (void**)&arr2_d, n * sizeof(float) );
        hipMalloc( (void**)&result_d, n * sizeof(float) );
        hipMemcpy( arr1_d, arr1->GetData(), n * sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( arr2_d, arr2->GetData(), n * sizeof(float), hipMemcpyHostToDevice );
        ElementWise_Kernel<<< n, 1 >>>( arr1_d, arr2_d, result_d, static_cast<int>(op) );
        hipMemcpy( result->GetData(), result_d, n * sizeof(float), hipMemcpyDeviceToHost );
        hipFree(arr1_d);
        hipFree(arr2_d);
        hipFree(result_d);
    }
}