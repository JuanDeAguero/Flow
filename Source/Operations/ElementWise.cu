#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#pragma once

#include "Flow/NArrayCore.h"

#include <hip/hip_runtime.h>

namespace Flow
{// CUDA Kernel
__global__ void ElementWise_Kernel(float* arr1, float* arr2, float* result, int totalSize, NArrayCore::Operation op) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < totalSize) {
        switch (op) {
            case NArrayCore::Operation::ADD:
                result[tid] = arr1[tid] + arr2[tid];
                break;
            case NArrayCore::Operation::MUL:
                result[tid] = arr1[tid] * arr2[tid];
                break;
            // Add other operations as needed...
        }
    }
}

// CUDA error checking
void checkCudaErrors(hipError_t status) {
    if (status != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(status));
        exit(1);
    }
}

// CUDA wrapper function
void ElementWise_CUDA(NArrayCore* arr1, NArrayCore* arr2, NArrayCore* result, NArrayCore::Operation op) {
    int totalSize = SizeFromShape(arr1->GetShape());

    if (totalSize != SizeFromShape(arr2->GetShape())) {
        fprintf(stderr, "Arrays shapes don't match for element-wise operations.\n");
        exit(1);
    }

    // Allocate GPU memory
    float* d_arr1;
    float* d_arr2;
    float* d_result;
    checkCudaErrors(hipMalloc(&d_arr1, totalSize * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_arr2, totalSize * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_result, totalSize * sizeof(float)));

    // Transfer data to GPU
    checkCudaErrors(hipMemcpy(d_arr1, arr1->Get().data(), totalSize * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_arr2, arr2->Get().data(), totalSize * sizeof(float), hipMemcpyHostToDevice));

    // Define launch configuration
    const int blockSize = 256; // This can be optimized for specific hardware
    const int gridSize = (totalSize + blockSize - 1) / blockSize;

    // Launch the kernel
    ElementWise_Kernel<<<gridSize, blockSize>>>(d_arr1, d_arr2, d_result, totalSize, op);
    checkCudaErrors(hipGetLastError()); // Check for errors in kernel launch

    // Synchronize threads to ensure all have completed
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer result back to host
    checkCudaErrors(hipMemcpy(result->Get().data(), d_result, totalSize * sizeof(float), hipMemcpyDeviceToHost));

    // Clean up GPU memory
    checkCudaErrors(hipFree(d_arr1));
    checkCudaErrors(hipFree(d_arr2));
    checkCudaErrors(hipFree(d_result));
}

}