#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void ReLU_Kernel( float* result )
{
    int i = blockIdx.x;
    Flow::AtomicMax_Device( &result[i], 0.0f );
}

Flow::NArrayCore* Flow::ReLU( NArrayCore* arr )
{
    int n = SizeFromShape(arr->GetShape());
    float* result_d;
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), n * sizeof(float), hipMemcpyDeviceToDevice );
    ReLU_Kernel<<< n, 1 >>>(result_d);
    hipDeviceSynchronize();
    return new NArrayCore( arr->GetShape(), result_d, { arr }, NArrayCore::Operation::RELU );
}

__global__
void BackwardReLU_Kernel( float* gradient, float* operand, float* operandGradient )
{
    int i = blockIdx.x;
    float grad = ( operand[i] > 0.0f ) ? gradient[i] : 0.0f;
    atomicAdd( &operandGradient[i], grad );
}

void Flow::NArrayCore::BackwardReLU()
{
    int n = SizeFromShape(Shape);
    BackwardReLU_Kernel<<< n, 1 >>>( Gradient->GetData(), Operands[0]->GetData(), Operands[0]->GetGradient()->GetData() );
    hipDeviceSynchronize();
}