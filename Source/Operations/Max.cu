#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <limits>

#include "CUDA.cuh"
#include "Flow/NArray.h"

__global__
void Max_Kernel( float* arr, int* arrShape, int arrShapeSize, int dim, float* result,
    int* resultShape, int resultShapeSize )
{
    int i = blockIdx.x;
    int multiIndex[10];
    Flow::FlatToMultiIndex_Device( i, arrShape, arrShapeSize, multiIndex );
    multiIndex[dim] = 0;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, resultShape, resultShapeSize );
    Flow::AtomicMax_Device( &result[flatIndex], arr[i] );
}

NARRAY Flow::Max( NARRAY arr, int dim )
{
    int n = SizeFromShape(arr->GetShape());
    vector<int> resultShape = arr->GetShape();
    resultShape[dim] = 1;
    vector<float> resultData( SizeFromShape(resultShape), numeric_limits<float>::min() );
    int* arrShape_d;
    float* result_d;
    int* resultShape_d;
    hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMalloc( (void**)&resultShape_d, resultShape.size() * sizeof(int) );
    hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( result_d, resultData.data(), SizeFromShape(resultShape) * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( resultShape_d, resultShape.data(), resultShape.size() * sizeof(int),
        hipMemcpyHostToDevice );
    Max_Kernel<<< n, 1 >>>( arr->GetData(), arrShape_d, arr->GetShape().size(), dim, result_d,
        resultShape_d, resultShape.size() );
    hipDeviceSynchronize();
    hipFree(arrShape_d);
    hipFree(resultShape_d);
    NARRAY result = Create( resultShape, result_d, { arr }, NArray::Operation::MAX );
    result->MaxDim = dim;
    return result;
}

__global__
void BackwardMax_Kernel( float* arr, int* shape, int shapeSize, float* gradient, float* operand,
    int* operandShape, int operandShapeSize, float* operandGradient, int dim )
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int multiIndex[10];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, multiIndex );
    multiIndex[dim] = j;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, operandShape, operandShapeSize );
    if ( operand[flatIndex] == arr[i] )
        atomicAdd( &operandGradient[flatIndex], gradient[i] );
}

void Flow::NArray::BackwardMax()
{
    int n = SizeFromShape(Shape);
    int* shape_d;
    int* operandShape_d;
    hipMalloc( (void**)&shape_d, Shape.size() * sizeof(int) );
    hipMalloc( (void**)&operandShape_d, Operands[0]->GetShape().size() * sizeof(int) );
    hipMemcpy( shape_d, GetShapeData(), Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( operandShape_d, Operands[0]->GetShapeData(),
        Operands[0]->GetShape().size() * sizeof(int), hipMemcpyHostToDevice );
    int maxDimSize = Operands[0]->GetShape()[MaxDim];
    dim3 gridDims( n, maxDimSize );
    BackwardMax_Kernel<<< gridDims, 1 >>>( GetData(), shape_d, Shape.size(), Gradient->GetData(),
        Operands[0]->GetData(), operandShape_d, Operands[0]->GetShape().size(),
        Operands[0]->GetGradient()->GetData(), MaxDim );
    hipDeviceSynchronize();
    hipFree(shape_d);
    hipFree(operandShape_d);
}