#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArray.h"

NARRAY Flow::Squeeze( NARRAY arr, int dim )
{
    vector<int> resultShape = arr->GetShape();
    resultShape.erase( resultShape.begin() + dim );
    float* result_d;
    hipMalloc( (void**)&result_d, SizeFromShape(arr->GetShape()) * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), SizeFromShape(arr->GetShape()) * sizeof(float),
        hipMemcpyDeviceToDevice );
    return Create( resultShape, result_d, { arr }, NArray::Operation::SQUEEZE );
}

__global__
void BackwardSqueeze_Kernel( float* gradient, float* operandGradient )
{
    int i = blockIdx.x;
    atomicAdd( &operandGradient[i], gradient[i] );
}

void Flow::NArray::BackwardSqueeze()
{
    int n = SizeFromShape(Shape);
    BackwardSqueeze_Kernel<<< n, 1 >>>( Gradient->GetData(),
        Operands[0]->GetGradient()->GetData() );
    hipDeviceSynchronize();
}