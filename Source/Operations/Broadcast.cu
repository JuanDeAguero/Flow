#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <stdexcept>

#include "CUDA.cuh"
#include "Flow/NArray.h"

std::vector<int> Flow::BroadcastShapes( vector<int> shape1, vector<int> shape2 )
{
    int maxDims = max( shape1.size(), shape2.size() );
    while ( shape1.size() < maxDims ) shape1.insert( shape1.begin(), 1 );
    while ( shape2.size() < maxDims ) shape2.insert( shape2.begin(), 1 );
    vector<int> shape(maxDims);
    for ( int i = 0; i < maxDims; i++ )
    {
        if ( shape1[i] == shape2[i] ) shape[i] = shape1[i];
        else if ( shape1[i] == 1 ) shape[i] = shape2[i];
        else if ( shape2[i] == 1 ) shape[i] = shape1[i];
        else throw runtime_error("Incompatible shapes for broadcast!");
    }
    return shape;
}

__global__
void Broadcast_Kernel( float* arr, int* arrShape, int arrShapeSize, int* shape, int shapeSize,
    float* result )
{
    int i = blockIdx.x;
    int multiIndex[MAX_DIMS];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, multiIndex );
    int originalCoords[MAX_DIMS];
    for ( int j = 0; j < arrShapeSize; j++ )
    {
        int coord = multiIndex[ shapeSize - arrShapeSize + j ];
        if ( arrShape[j] == 1 ) coord = 0;
        originalCoords[j] = coord;
    }
    int flatIndex = Flow::MultiToFlatIndex_Device( originalCoords, arrShape, arrShapeSize );
    result[i] = arr[flatIndex];
}

NARRAY Flow::Broadcast( NARRAY arr, vector<int> shape )
{
    int n = SizeFromShape(shape);
    int* arrShape_d;
    int* shape_d;
    float* result_d;
    hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
    hipMalloc( (void**)&shape_d, shape.size() * sizeof(int) );
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( shape_d, shape.data(), shape.size() * sizeof(int), hipMemcpyHostToDevice );
    Broadcast_Kernel<<< n, 1 >>>( arr->GetData(), arrShape_d, arr->GetShape().size(), shape_d,
        shape.size(), result_d );
    hipFree(arrShape_d);
    hipFree(shape_d);
    return Create( shape, result_d, { arr }, NArray::Operation::BROADCAST );
}

__global__
void BackwardBroadcast_Kernel( float* gradient, int* shape, int shapeSize, int* operandShape,
    int operandShapeSize, float* operandGradient )
{
    int i = blockIdx.x;
    int multiIndex[MAX_DIMS];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, multiIndex );
    int operandCoords[MAX_DIMS];
    for ( int j = 0; j < operandShapeSize; j++ )
    {
        int coord = multiIndex[ shapeSize - operandShapeSize + j ];
        if ( operandShape[j] == 1 ) operandCoords[j] = 0;
        else operandCoords[j] = coord;
    }
    int operandIndex = Flow::MultiToFlatIndex_Device( operandCoords, operandShape,
        operandShapeSize );
    atomicAdd( &operandGradient[operandIndex], gradient[i] );
}

__host__
void Flow::NArray::BackwardBroadcast()
{
    int n = SizeFromShape(Gradient->GetShape());
    int* shape_d;
    int* operandShape_d;
    hipMalloc( (void**)&shape_d, Shape.size() * sizeof(int) );
    hipMalloc( (void**)&operandShape_d, Operands[0]->Shape.size() * sizeof(int) );
    hipMemcpy( shape_d, GetShapeData(), Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( operandShape_d, Operands[0]->GetShapeData(),
        Operands[0]->Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    BackwardBroadcast_Kernel<<< n, 1 >>>( Gradient->GetData(), shape_d, Shape.size(),
        operandShape_d, Operands[0]->Shape.size(), Operands[0]->Gradient->GetData() );
    hipFree(shape_d);
    hipFree(operandShape_d);
}