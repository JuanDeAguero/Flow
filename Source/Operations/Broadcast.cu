#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Broadcast_Kernel( float* arr, int* arrShape, int arrShapeSize, int* shape, int shapeSize, float* result )
{
    int i = blockIdx.x;
    int position[10];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, position );
    int originalCoords[10];
    for ( int j = 0; j < arrShapeSize; j++ )
    {
        int coord = position[ shapeSize - arrShapeSize + j ];
        if ( arrShape[j] == 1 ) coord = 0;
        originalCoords[j] = coord;
    }
    int flatIndex = Flow::MultiToFlatIndex_Device( originalCoords, arrShape, arrShapeSize );
    result[i] = arr[flatIndex];
}

namespace Flow
{
    __host__
    NArrayCore* Broadcast_CUDA( NArrayCore* arr, vector<int> shape )
    {
        int n = SizeFromShape(shape);
        float* arr_d;
        int* arrShape_d;
        int* shape_d;
        float* result_d;
        hipMalloc( (void**)&arr_d, arr->Get().size() * sizeof(float) );
        hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
        hipMalloc( (void**)&shape_d, shape.size() * sizeof(int) );
        hipMalloc( (void**)&result_d, n * sizeof(float) );
        hipMemcpy( arr_d, arr->GetData(), arr->Get().size() * sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( shape_d, shape.data(), shape.size() * sizeof(int), hipMemcpyHostToDevice );
        Broadcast_Kernel<<< n, 1 >>>( arr_d, arrShape_d, arr->GetShape().size(), shape_d, shape.size(), result_d );
        vector<float> resultData(n);
        hipMemcpy( resultData.data(), result_d, n * sizeof(float), hipMemcpyDeviceToHost );
        hipFree(arr_d);
        hipFree(arrShape_d);
        hipFree(shape_d);
        hipFree(result_d);
        return new NArrayCore( shape, resultData, { arr }, NArrayCore::Operation::BROADCAST );
    }
}

__global__
void BackwardBroadcast_Kernel()
{
    int i = blockIdx.x;

}

__host__
void Flow::NArrayCore::BackwardBroadcast_CUDA()
{

}