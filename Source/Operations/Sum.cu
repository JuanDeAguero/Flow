#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <limits>

#include "CUDA.cuh"
#include "Flow/NArray.h"

__global__
void Sum_Kernel( float* arr, int* arrShape, int arrShapeSize, int dim, float* result,
    int* resultShape, int resultShapeSize )
{
    int i = blockIdx.x;
    int multiIndex[10];
    Flow::FlatToMultiIndex_Device( i, arrShape, arrShapeSize, multiIndex );
    multiIndex[dim] = 0;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, resultShape, resultShapeSize );
    atomicAdd( &result[flatIndex], arr[i] );
}

NARRAY Flow::Sum( NARRAY arr, int dim )
{
    int n = SizeFromShape(arr->GetShape());
    vector<int> resultShape = arr->GetShape();
    resultShape[dim] = 1;
    vector<float> resultData( SizeFromShape(resultShape), numeric_limits<float>::min() );
    int* arrShape_d;
    float* result_d;
    int* resultShape_d;
    hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMalloc( (void**)&resultShape_d, resultShape.size() * sizeof(int) );
    hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( result_d, resultData.data(), SizeFromShape(resultShape) * sizeof(int),
        hipMemcpyHostToDevice );
    hipMemcpy( resultShape_d, resultShape.data(), resultShape.size() * sizeof(int),
        hipMemcpyHostToDevice );
    Sum_Kernel<<< n, 1 >>>( arr->GetData(), arrShape_d, arr->GetShape().size(), dim, result_d,
        resultShape_d, resultShape.size() );
    hipDeviceSynchronize();
    hipFree(arrShape_d);
    hipFree(resultShape_d);
    NARRAY result = NArray::Create( resultShape, result_d, { arr }, NArray::Operation::SUM );
    result->SumDim = dim;
    return result;
}

__global__
void BackwardSum_Kernel( float* arr, int* shape, int shapeSize, float* gradient, float* operand,
    int* operandShape, int operandShapeSize, float* operandGradient, int dim )
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int multiIndex[10];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, multiIndex );
    multiIndex[dim] = j;
    int flatIndex = Flow::MultiToFlatIndex_Device( multiIndex, operandShape, operandShapeSize );
    atomicAdd( &operandGradient[flatIndex], gradient[i] );
}

void Flow::NArrayCore::BackwardSum()
{
    int n = SizeFromShape(Shape);
    int* shape_d;
    int* operandShape_d;
    hipMalloc( (void**)&shape_d, Shape.size() * sizeof(int) );
    hipMalloc( (void**)&operandShape_d, Operands[0]->GetShape().size() * sizeof(int) );
    hipMemcpy( shape_d, GetShapeData(), Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( operandShape_d, Operands[0]->GetShapeData(),
        Operands[0]->GetShape().size() * sizeof(int), hipMemcpyHostToDevice );
    int maxDimSize = Operands[0]->GetShape()[SumDim];
    dim3 gridDims( n, maxDimSize );
    BackwardSum_Kernel<<< gridDims, 1 >>>( GetData(), shape_d, Shape.size(), Gradient->GetData(),
        Operands[0]->GetData(), operandShape_d, Operands[0]->GetShape().size(),
        Operands[0]->GetGradient()->GetData(), SumDim );
    hipDeviceSynchronize();
    hipFree(shape_d);
    hipFree(operandShape_d);
}