#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <cmath>

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Pow_Kernel( float* result, float exponent )
{
    int i = blockIdx.x;
    result[i] = pow( result[i], exponent );
}

Flow::NArrayCore* Flow::Pow( NArrayCore* arr, float exponent )
{
    int n = SizeFromShape(arr->GetShape());
    float* result_d;
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), n * sizeof(float), hipMemcpyDeviceToDevice );
    Pow_Kernel<<< n, 1 >>>( result_d, exponent );
    hipDeviceSynchronize();
    NArrayCore* result = new NArrayCore( arr->GetShape(), result_d, { arr }, NArrayCore::Operation::POW );
    result->Exponent = exponent;
    return result;
}

__global__
void BackwardPow_Kernel( float* gradient, float* operand, float* operandGradient, float exponent )
{
    int i = blockIdx.x;
    float grad = gradient[i] * exponent * pow( operand[i], exponent - 1);
    atomicAdd( &operandGradient[i], grad );
}

void Flow::NArrayCore::BackwardPow()
{
    int n = SizeFromShape(Shape);
    BackwardPow_Kernel<<< n, 1 >>>( Gradient->GetData(), Operands[0]->GetData(), Operands[0]->GetGradient()->GetData(), Exponent );
    hipDeviceSynchronize();
}