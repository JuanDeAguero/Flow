#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <cmath>

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Log_Kernel( float* result )
{
    int i = blockIdx.x;
    result[i] = log(result[i]);
}

Flow::NArrayCore* Flow::Log( NArrayCore* arr )
{
    int n = SizeFromShape(arr->GetShape());
    float* result_d;
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), n * sizeof(float), hipMemcpyDeviceToDevice );
    Log_Kernel<<< n, 1 >>>(result_d);
    return new NArrayCore( arr->GetShape(), result_d, { arr }, NArrayCore::Operation::LOG );
}

__global__
void BackwardLog_Kernel( float* gradient, float* operand, float* operandGradient )
{
    int i = blockIdx.x;
    float grad = gradient[i] / operand[i];
    operandGradient[i] += grad;
}

void Flow::NArrayCore::BackwardLog()
{
    int n = SizeFromShape(Shape);
    BackwardLog_Kernel<<< n, 1 >>>( Gradient->GetData(), Operands[0]->GetData(), Operands[0]->GetGradient()->GetData() );
}