#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void BackwardMul_Kernel( float* gradient, float* operand1, float* operandGradient1, float* operand2, float* operandGradient2 )
{
    int i = blockIdx.x;
    operandGradient1[i] += operand2[i] * gradient[i];
    operandGradient2[i] += operand1[i] * gradient[i];
}

__host__
void Flow::NArrayCore::BackwardMul_CUDA()
{
    int n = Gradient->Data.size();
    float* gradient_d;
    float* operand1_d;
    float* operandGradient1_d;
    float* operand2_d;
    float* operandGradient2_d;
    hipMalloc( (void**)&gradient_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient1_d, n * sizeof(float) );
    hipMalloc( (void**)&operand1_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient2_d, n * sizeof(float) );
    hipMalloc( (void**)&operand2_d, n * sizeof(float) );
    hipMemcpy( gradient_d, Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient1_d, Operands[0]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operand1_d, Operands[0]->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient2_d, Operands[1]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operand2_d, Operands[1]->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    BackwardMul_Kernel<<< n, 1 >>>( gradient_d, operand1_d, operandGradient1_d, operand2_d, operandGradient2_d );
    hipMemcpy( Operands[0]->Gradient->Data.data(), operandGradient1_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Operands[1]->Gradient->Data.data(), operandGradient2_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipFree(gradient_d);
    hipFree(operand1_d);
    hipFree(operandGradient1_d);
    hipFree(operand2_d);
    hipFree(operandGradient2_d);
}