#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void BackwardAdd_Kernel( float* gradient, float* operandGradient1, float* operandGradient2 )
{
    int i = blockIdx.x;
    operandGradient1[i] += gradient[i];
    operandGradient2[i] += gradient[i];
}

__host__
void Flow::NArrayCore::BackwardAdd_CUDA()
{
    int n = Gradient->Data.size();
    float* gradient_d;
    float* operandGradient1_d;
    float* operandGradient2_d;
    hipMalloc( (void**)&gradient_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient1_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient2_d, n * sizeof(float) );
    hipMemcpy( gradient_d, Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient1_d, Operands[0]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient2_d, Operands[1]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    BackwardAdd_Kernel<<< n, 1 >>>( gradient_d, operandGradient1_d, operandGradient2_d );
    hipMemcpy( Operands[0]->Gradient->Data.data(), operandGradient1_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Operands[1]->Gradient->Data.data(), operandGradient2_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipFree(gradient_d);
    hipFree(operandGradient1_d);
    hipFree(operandGradient2_d);
}