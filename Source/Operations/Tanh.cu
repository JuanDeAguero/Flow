#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include <cmath>

#include "CUDA.cuh"
#include "Flow/NArray.h"

__global__
void Tanh_Kernel( float* result )
{
    int i = blockIdx.x;
    result[i] = tanh(result[i]);
}

NARRAY Flow::Tanh( NARRAY arr )
{
    int n = SizeFromShape(arr->GetShape());
    float* result_d;
    hipMalloc( (void**)&result_d, n * sizeof(float) );
    hipMemcpy( result_d, arr->GetData(), n * sizeof(float), hipMemcpyDeviceToDevice );
    Tanh_Kernel<<< n, 1 >>>(result_d);
    hipDeviceSynchronize();
    return NArray::Create( arr->GetShape(), result_d, { arr }, NArray::Operation::TANH );
}

__global__
void BackwardTanh_Kernel( float* gradient, float* operand, float* operandGradient )
{
    int i = blockIdx.x;
    float value = tanh(operand[i]);
    float grad = gradient[i] * ( 1 - value * value );
    atomicAdd( &operandGradient[i], grad );
}

void Flow::NArrayCore::BackwardTanh()
{
    int n = SizeFromShape(Shape);
    BackwardTanh_Kernel<<< n, 1 >>>( Gradient->GetData(), Operands[0]->GetData(),
        Operands[0]->GetGradient()->GetData() );
    hipDeviceSynchronize();
}