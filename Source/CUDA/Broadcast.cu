#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Broadcast_Kernel( float* arr, int* arrShape, int arrShapeSize, int* shape, int shapeSize, float* result )
{
    int i = blockIdx.x;
    int position[10];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, position );
    int originalCoords[10];
    for ( int j = 0; j < arrShapeSize; j++ )
    {
        int coord = position[ shapeSize - arrShapeSize + j ];
        if ( arrShape[j] == 1 ) coord = 0;
        originalCoords[j] = coord;
    }
    int flatIndex = Flow::MultiToFlatIndex_Device( originalCoords, arrShape, arrShapeSize );
    result[i] = arr[flatIndex];
}

namespace Flow
{
    __host__
    NArrayCore* Broadcast_CUDA( NArrayCore* arr, vector<int> shape )
    {
        int n = SizeFromShape(shape);
        float* arr_d;
        int* arrShape_d;
        int* shape_d;
        float* result_d;
        hipMalloc( (void**)&arr_d, arr->Get().size() * sizeof(float) );
        hipMalloc( (void**)&arrShape_d, arr->GetShape().size() * sizeof(int) );
        hipMalloc( (void**)&shape_d, shape.size() * sizeof(int) );
        hipMalloc( (void**)&result_d, n * sizeof(float) );
        hipMemcpy( arr_d, arr->GetData(), arr->Get().size() * sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( arrShape_d, arr->GetShapeData(), arr->GetShape().size() * sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( shape_d, shape.data(), shape.size() * sizeof(int), hipMemcpyHostToDevice );
        Broadcast_Kernel<<< n, 1 >>>( arr_d, arrShape_d, arr->GetShape().size(), shape_d, shape.size(), result_d );
        vector<float> resultData(n);
        hipMemcpy( resultData.data(), result_d, n * sizeof(float), hipMemcpyDeviceToHost );
        hipFree(arr_d);
        hipFree(arrShape_d);
        hipFree(shape_d);
        hipFree(result_d);
        return new NArrayCore( shape, resultData, { arr }, NArrayCore::Operation::BROADCAST );
    }
}

__global__
void BackwardBroadcast_Kernel_A( int* shape, int shapeSize, int* operandShape, int operandShapeSize, float* newOperandGradient, float* gradient )
{
    int i = blockIdx.x;
    int position[10];
    Flow::FlatToMultiIndex_Device( i, shape, shapeSize, position );
    int operandCoords[10];
    for ( int j = 0; j < operandShapeSize; j++ )
    {
        int coord = position[ shapeSize - operandShapeSize + j ];
        if ( operandShape[j] == 1 ) coord = 0;
        operandCoords[j] = coord;
    }
    int operandIndex = Flow::MultiToFlatIndex_Device( operandCoords, operandShape, operandShapeSize );
    atomicAdd( &newOperandGradient[operandIndex], gradient[i] );
}

__global__
void BackwardBroadcast_Kernel_B( float* operandGradient, float* newOperandGradient )
{
    int i = blockIdx.x;
    operandGradient[i] += newOperandGradient[i];
}

__host__
void Flow::NArrayCore::BackwardBroadcast_CUDA()
{
    vector<float> newOperandGradient( Operands[0]->Data.size(), 0.0f );
    int n = Gradient->Data.size();
    int* shape_d;
    int* operandShape_d;
    float* newOperandGradient_d;
    float* gradient_d;
    hipMalloc( (void**)&shape_d, Shape.size() * sizeof(int) );
    hipMalloc( (void**)&operandShape_d, Operands[0]->Shape.size() * sizeof(int) );
    hipMalloc( (void**)&newOperandGradient_d, newOperandGradient.size() * sizeof(float) );
    hipMalloc( (void**)&gradient_d, Gradient->Get().size() * sizeof(float) );
    hipMemcpy( shape_d, GetShapeData(), Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( operandShape_d, Operands[0]->GetShapeData(), Operands[0]->Shape.size() * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( newOperandGradient_d, newOperandGradient.data(), newOperandGradient.size() * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( gradient_d, Gradient->GetData(), Gradient->Get().size() * sizeof(float), hipMemcpyHostToDevice );
    BackwardBroadcast_Kernel_A<<< n, 1 >>>( shape_d, Shape.size(), operandShape_d, Operands[0]->Shape.size(), newOperandGradient_d, gradient_d );
    n = Operands[0]->Gradient->Data.size();
    float* operandGradient_d;
    hipMalloc( (void**)&operandGradient_d, Operands[0]->Gradient->Get().size() * sizeof(float) );
    hipMemcpy( operandGradient_d, Operands[0]->Gradient->GetData(), Operands[0]->Gradient->Get().size() * sizeof(float), hipMemcpyHostToDevice );
    BackwardBroadcast_Kernel_B<<< n, 1 >>>( operandGradient_d, newOperandGradient_d );
    hipMemcpy( Operands[0]->Gradient->GetData(), operandGradient_d, Operands[0]->Gradient->Get().size() * sizeof(float), hipMemcpyDeviceToHost );
    hipFree(shape_d);
    hipFree(operandShape_d);
    hipFree(newOperandGradient_d);
    hipFree(gradient_d);
    hipFree(operandGradient_d);
}