#include "hip/hip_runtime.h"
// Copyright (c) 2023 Juan M. G. de Agüero

#include "CUDA.cuh"
#include "Flow/NArrayCore.h"

__global__
void Mul_Kernel( float* arr1, float* arr2, float* result )
{
    int i = blockIdx.x;
    result[i] = arr1[i] * arr2[i];
}
    
namespace Flow
{
    __host__
    NArrayCore* Mul_CUDA( NArrayCore* arr1, NArrayCore* arr2 )
    {
        int n = arr1->Get().size();
        float* arr1_d;
        float* arr2_d;
        float* result_d;
        hipMalloc( (void**)&arr1_d, n * sizeof(float) );
        hipMalloc( (void**)&arr2_d, n * sizeof(float) );
        hipMalloc( (void**)&result_d, n * sizeof(float) );
        hipMemcpy( arr1_d, arr1->GetData(), n * sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( arr2_d, arr2->GetData(), n * sizeof(float), hipMemcpyHostToDevice );
        Mul_Kernel<<< n, 1 >>>( arr1_d, arr2_d, result_d );
        vector<float> resultData(n);
        hipMemcpy( resultData.data(), result_d, n * sizeof(float), hipMemcpyDeviceToHost );
        hipFree(arr1_d);
        hipFree(arr2_d);
        hipFree(result_d);
        return new NArrayCore( arr1->GetShape(), resultData, { arr1, arr2 }, NArrayCore::Operation::MUL );
    }
}

__global__
void BackwardMul_Kernel( float* gradient, float* operand1, float* operandGradient1, float* operand2, float* operandGradient2 )
{
    int i = blockIdx.x;
    operandGradient1[i] += operand2[i] * gradient[i];
    operandGradient2[i] += operand1[i] * gradient[i];
}

__host__
void Flow::NArrayCore::BackwardMul_CUDA()
{
    int n = Gradient->Data.size();
    float* gradient_d;
    float* operand1_d;
    float* operandGradient1_d;
    float* operand2_d;
    float* operandGradient2_d;
    hipMalloc( (void**)&gradient_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient1_d, n * sizeof(float) );
    hipMalloc( (void**)&operand1_d, n * sizeof(float) );
    hipMalloc( (void**)&operandGradient2_d, n * sizeof(float) );
    hipMalloc( (void**)&operand2_d, n * sizeof(float) );
    hipMemcpy( gradient_d, Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient1_d, Operands[0]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operand1_d, Operands[0]->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operandGradient2_d, Operands[1]->Gradient->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( operand2_d, Operands[1]->Data.data(), n * sizeof(float), hipMemcpyHostToDevice );
    BackwardMul_Kernel<<< n, 1 >>>( gradient_d, operand1_d, operandGradient1_d, operand2_d, operandGradient2_d );
    hipMemcpy( Operands[0]->Gradient->Data.data(), operandGradient1_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( Operands[1]->Gradient->Data.data(), operandGradient2_d, n * sizeof(float), hipMemcpyDeviceToHost );
    hipFree(gradient_d);
    hipFree(operand1_d);
    hipFree(operandGradient1_d);
    hipFree(operand2_d);
    hipFree(operandGradient2_d);
}